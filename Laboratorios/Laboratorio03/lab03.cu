#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

#define MATRIX_SIZE 2
#define BLOCK_SIZE 16

// Kernel for matrix addition in CUDA
__global__ void matrixAddKernel(int* a, int* b, int* c, int size) {
    int threadX = threadIdx.x;
    int threadY = threadIdx.y;

    int blockX = blockIdx.x;
    int blockY = blockIdx.y;

    int dimX = blockDim.x;
    int dimY = blockDim.y;

    int globalIdx = blockX * dimX + threadX;
    int globalIdy = blockY * dimY + threadY;

    int globalId = (globalIdy * blockDim.x * gridDim.x) + globalIdx;

    c[globalId] = a[globalId] + b[globalId];
}

// Function to initialize a matrix with random values
void initializeMatrix(int* matrix, int col, int row, int number) {
    for (long int i = 0; i < row * col; i++) {
            matrix[i] = number;
    }
}

void printMatrix(int* matrix, int size) {
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            printf("Matrix Value: %d \n", matrix[i * size + j]);
        }
    }
}

int main() {
    // Host matrices variables
    int* hostMatrixA, * hostMatrixB, * hostMatrixC;
    int* hostMatrixA2, * hostMatrixB2, * hostMatrixC2;

    // Device matrices variables
    int* deviceMatrixA, * deviceMatrixB, * deviceMatrixC;
    int* deviceMatrixA2, * deviceMatrixB2, * deviceMatrixC2;
    
    // Matrix size
    int col = 100;
    int row = 100;

    int col2 = 2;
    int row2 = 2;
    
    // Size in bytes
    long int bytes = col * row * sizeof(int);
    long int bytes2 = col2 * row2 * sizeof(int);

    // Memory allocation on the host
    hostMatrixA = (int*)malloc(bytes);
    hostMatrixB = (int*)malloc(bytes);
    hostMatrixC = (int*)malloc(bytes);

    hostMatrixA2 = (int*)malloc(bytes2);
    hostMatrixB2 = (int*)malloc(bytes2);
    hostMatrixC2 = (int*)malloc(bytes2);

    // Initialize matrices hostMatrixA and hostMatrixB with random values
    initializeMatrix(hostMatrixA, row, col, 1);
    initializeMatrix(hostMatrixB, row, col, 2);

    // Initialize matrices hostMatrixA2 and hostMatrixB2 with random values
    initializeMatrix(hostMatrixA2, row2, col2, 1);
    initializeMatrix(hostMatrixB2, row2, col2, 2);

    // Allocate memory on the device
    hipMalloc(&deviceMatrixA, bytes);
    hipMalloc(&deviceMatrixB, bytes);
    hipMalloc(&deviceMatrixC, bytes);

    hipMalloc(&deviceMatrixA2, bytes2);
    hipMalloc(&deviceMatrixB2, bytes2);
    hipMalloc(&deviceMatrixC2, bytes2);

    // Copy data from host to device
    hipMemcpy(deviceMatrixA, hostMatrixA, bytes, hipMemcpyHostToDevice);
    hipMemcpy(deviceMatrixB, hostMatrixB, bytes, hipMemcpyHostToDevice);

    hipMemcpy(deviceMatrixA2, hostMatrixA2, bytes, hipMemcpyHostToDevice);
    hipMemcpy(deviceMatrixB2, hostMatrixB2, bytes, hipMemcpyHostToDevice);

    // Define block and grid dimensions
    dim3 blockSize(1, 1, 1);
    dim3 gridSize(1, 1, 1);

    // Measure execution time
    auto start = std::chrono::high_resolution_clock::now();

    // Launch the matrix addition kernel
    matrixAddKernel << <gridSize, blockSize >> > (deviceMatrixA, deviceMatrixB, deviceMatrixC, row);
    //matrixMultiplyKernel << <gridSize, blockSize >> > (deviceMatrixA2, deviceMatrixB2, deviceMatrixC2, 2);
    hipDeviceSynchronize();

    // Measure execution time
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float, std::milli> duration = end - start;
    std::cout << "Execution time: " << duration.count() << " ms" << std::endl;

    // Check for CUDA errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(error) << std::endl;
    }

    // Copy results from device to host
    hipMemcpy(hostMatrixC, deviceMatrixC, bytes, hipMemcpyDeviceToHost);

    hipMemcpy(hostMatrixC2, deviceMatrixC2, bytes, hipMemcpyDeviceToHost);

    // Check that the matrices are computed correctly
    //printMatrix(hostMatrixC, row);

    //printMatrix(hostMatrixC2, row2);

    // Free memory
    free(hostMatrixA);
    free(hostMatrixB);
    free(hostMatrixC);
    hipFree(deviceMatrixA);
    hipFree(deviceMatrixB);
    hipFree(deviceMatrixC);

    free(hostMatrixA2);
    free(hostMatrixB2);
    free(hostMatrixC2);
    hipFree(deviceMatrixA2);
    hipFree(deviceMatrixB2);
    hipFree(deviceMatrixC2);

    return 0;
}
