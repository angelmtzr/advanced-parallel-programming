#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void print_hello_cuda()
{
	printf("[DEVICE] ThreadIdx.x: %d\n", threadIdx.x);
}

int main() 
{
	print_hello_cuda << <2, 8 >> > ();
	return 0;
}